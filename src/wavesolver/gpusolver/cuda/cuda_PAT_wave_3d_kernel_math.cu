#include "hip/hip_runtime.h"
__device__ __forceinline__ Number_t pat_wave_3d_absortion(const Number_t pos, const Number_t dmin, const Number_t dmax, const Number_t strength, const Number_t width){
	Number_t d;
	Number_t pd = min(pos-dmin, dmax-pos);
	if(pd < width){
		d = (width-pd)/width;
		return strength*d*d;
	} else{
		return 0;
	}
}

__device__ __forceinline__ Number_t pat_wave_3d_vel_update(const Number_t idt, const Number_t absortion){
	return (idt - absortion/2.0f)/(idt + absortion/2.0f);
}

__device__ __forceinline__ Number_t pat_wave_3d_directional(const Number_t idt, const Number_t absortion){
	return (idt + absortion/2.0f);
}

__device__ __forceinline__ Number_t pat_wave_3d_pre_update(const Number_t idt, const Number_t absortion, const Number_t directional){
	return (idt - absortion/2.0f)/directional;
}

__device__ __forceinline__ Number_t pat_wave_3d_pre_divergence(const Number_t density, const Number_t c, const Number_t directional, const Number_t cellSize){
	return (-1.0f*density*(c/cellSize)*(c/directional));
}

__device__ __forceinline__ Number_t pat_wave_3d_gradient(const Number_t idt, const Number_t absortion, const Number_t cellSize, const Number_t density){
	return -1.0f/(density*cellSize*(idt + absortion/2.0f));
}

__device__ __forceinline__ Number_t pat_amplitude(const Number_t p1, const Number_t p2, const Number_t t1, const Number_t t2, const Number_t omega, const Number_t phase){
	Number_t cosi = sin(omega*t1 + phase);
	// if(cosi == 0.0f){
	// 	cosi = sin(omega*t2 + phase);
	// 	return abs(p2/cosi);
	// }
	return abs(p1/cosi);
}

__device__ __forceinline__ Number_t pat_phase(const Number_t p1, const Number_t p2, const Number_t t1, const Number_t t2, const Number_t omega){
	return atan2(p1*cos(omega*t2) - p2*cos(omega*t1), p2*sin(omega*t1) - p1*sin(omega*t2));
}

__device__ __forceinline__ Number_t Amn(int m, int n){
	int mm = abs(m);
    return n < mm ? 0 : sqrt(((mm+n+1.0f)*(n-mm+1.0f)) / ((2*n+1.0f)*(2*n+3.0f)));
}

__device__ __forceinline__ Number_t Bmn(int m, int n){
    return n < abs(m) ? 0 :
        (m >= 0 ? (sqrt(((n-m-1.0f)*(n-m)) / ((2*n-1.0f)*(2*n+1.0f))))
        		:-(sqrt(((n-m-1.0f)*(n-m)) / ((2*n-1.0f)*(2*n+1.0f))))
        		);
}

__device__ __forceinline__ Number_t bess(int n){
	return (n >= 0 ? bessel[n] : 0);
}

__device__ __forceinline__ void regular_basis(const int m, const int n, const Number_t phi, const int offm, const int offn, Number_t out[2], const Number_t legendre[3][3]){
	const Number_t sphi = sin(m*phi);
	const Number_t cphi = cos(m*phi);
	int kofm (m < 0 ? 2-offm : offm);
	const Number_t harm = ((n >= abs(m)) ? ((m&1) ? -(bess(n)*(legendre[kofm][offn]))
										     	  	   : (bess(n)*(legendre[kofm][offn]))
										     	       )
									     : 0
									     );

	out[0] = harm*cphi;
	out[1] = harm*sphi;
}

__device__ __forceinline__ void complexScaleAdd(const Number_t a, const Number_t b[2], Number_t out[2]){
	out[0] += a*b[0];
	out[1] += a*b[1];
}

__device__ __forceinline__ void complexMultAdd(const Number_t a[2], const Number_t b[2], Number_t out[2]){
	out[0] += a[0]*b[0]-a[1]*b[1];
	out[1] += a[0]*b[1]+a[1]*b[0];
}

__device__ __forceinline__ void complexAdd(const Number_t a[2], const Number_t b[2], Number_t out[2]){
	out[0] += a[0]+b[0];
	out[1] += a[1]+b[1];
}

__device__ __forceinline__ Number_t normalize_angle_2pi(Number_t ang){
	if(isnan(ang)) return 0;
	const Number_t PI = acos(-1.0f);
	ang = fmod(ang, 2*PI);
	if(ang < 0) ang += 2*PI;
	return ang;
}

__device__ __forceinline__ Number_t normalize_angle_pi(Number_t ang){
	const Number_t PI = acos(-1.0f);
	ang = fmod(ang+PI, 2*PI);
	if(ang < 0) ang += 2*PI;
	return ang-PI;
}

//K is the weight for a
//Assumes that a and b are in [-pi, pi]
__device__ __forceinline__ Number_t interpolate_angles(Number_t a, Number_t b, Number_t k){
	const Number_t PI = acos(-1.0f);
	Number_t diff = a-b;
	if(diff < 0){
		Number_t t = a;
		a = b;
		b = t;
		k = 1-k;
		diff = -diff;
	}
	diff = (diff > PI ? diff - 2*PI
					  : diff);

	return normalize_angle_pi(b + diff*k);
}